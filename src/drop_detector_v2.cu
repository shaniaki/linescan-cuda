#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <hip/hip_runtime.h>
#include <assert.h>

#include "globals.hpp"

////////////////////////////////////////////////////////////////////////////////
// export the callable function
extern "C"
void compute_v2(float* reference,
				float* input_image,
				aoi* aoi_coordinates,
				float* parallelCoeffs,
				int* parallelSW,
				unsigned int image_height,
				unsigned int image_width);

__device__
void thread_allocator_v2(float* input_image,
						aoi* aoi_coordinates,
						unsigned int image_width,
						unsigned int sn,
						aoi* thread_state,
						float* image_parts)
{
	// thread allocator
	unsigned int ta = blockIdx.x;
	/// output decoding
	int index = 0;
	for (int taod = (thread_state + ta)->start;
			taod <= (thread_state + ta)->end; taod++) {
		image_parts[(image_width / N) * ta + index] =
				input_image[(sn * image_width) + taod];
		index++;
	}
	image_parts[(image_width / N) * ta + index] = -1; // to terminate the image part
	/// next state
	if ((aoi_coordinates + sn * N + ta)->start != -1)
		thread_state[ta] = aoi_coordinates[sn * N + ta];
}

__device__
float preproc_image_v2(float* image_parts, unsigned int image_width)
{
	unsigned int noz = blockIdx.x;
	float imgPreproc2DDFA = 0;
	float* imgpre = image_parts + (image_width / N) * noz;
	while (*(imgpre) != -1) {
		imgPreproc2DDFA += WHITE_VALUE - *(imgpre);
		imgpre++;
	}
	return imgPreproc2DDFA;
}

__device__
void auto_correlate_v2(float imgPreproc2DDFA,
						int* parallelSW,
						unsigned int sn,
						float* ac_samples,
						int* ac_sw,
						int ac_ignore_it,
						float* ac_sampWin,
						float* autoCorrToCombSubMul)
{
	unsigned int noz = blockIdx.x;
	unsigned int thx = threadIdx.x;
	//// output decoding
	if (thx<2*(*ac_sw))
		ac_sampWin[thx] = ac_samples[noz*(BUFFER_SIZE+1) + ac_ignore_it + thx];
	if (thx==0)
		ac_sampWin[2*(*ac_sw)] = ac_samples[noz*(BUFFER_SIZE+1) + ac_ignore_it + 2*(*ac_sw)];
	__syncthreads();
	if (thx<2*ac_sw[noz])
		for (int c = 0; c <= ac_sw[noz] * 2; c++) {
			int d = c + (thx-ac_sw[noz]) + 1;
			int k = (thx-ac_sw[noz]) + ac_sw[noz];
			if ((d >= 0) && (d < ac_sw[noz] * 2))
				autoCorrToCombSubMul[ac_sw[noz] * 2 - k - 1] += ac_sampWin[c] * ac_sampWin[d];
		}
	__syncthreads();
	//// next state
	int ac_temp = parallelSW[sn * N + noz];
	if (ac_temp != -1)
		ac_sw[noz] = ac_temp;

	for (int i=0;i<BUFFER_SIZE;i++)
		ac_samples[noz*(BUFFER_SIZE+1)+i] = ac_samples[noz*(BUFFER_SIZE+1)+(i+1)];
	ac_samples[noz*(BUFFER_SIZE+1) + BUFFER_SIZE] = imgPreproc2DDFA;
}

__device__
void cross_correlate_v2(int ac_ignore_it,
						int* ac_sw,
						float* ac_sampWin,
						float* cc_coefs,
						float* parallelCoeffs,
						unsigned int sn,
						float* xCorrToCombSubMul)
{
	unsigned int noz = blockIdx.x;
	unsigned int thx = threadIdx.x;
	//// output decoding
	__syncthreads();
	if (thx<2*ac_sw[noz])
		for (int c = 0; c <= ac_sw[noz] * 2; c++) {
			int d = c + (thx-ac_sw[noz]) + 1;
			int k = (thx-ac_sw[noz]) + ac_sw[noz];
			if ((d >= 0) && (d < ac_sw[noz] * 2))
				xCorrToCombSubMul[ac_sw[noz] * 2 - k - 1] += ac_sampWin[c]
				                                                        * cc_coefs[noz*(BUFFER_SIZE+1) + ac_ignore_it + d];
		}
	__syncthreads();
	//// next state
	float* cc_temp = parallelCoeffs + sn * N * BUFFER_SIZE
			+ noz * BUFFER_SIZE;
	if (*cc_temp != -1)
		if (thx<BUFFER_SIZE)
			cc_coefs[noz*(BUFFER_SIZE+1) + thx] = cc_temp[thx];
}

__device__
void submul_v2(float* combSubMulToCombAvgSub,
			float* xCorrToCombSubMul,
			float* autoCorrToCombSubMul,
			unsigned int win_size)
{
	int i=threadIdx.x;
	if (i < win_size)
		combSubMulToCombAvgSub[i] = (xCorrToCombSubMul[i]
		                           - autoCorrToCombSubMul[i]) / autoCorrToCombSubMul[i];
}

__device__
void avgsub_v2(float* combSubMulToCombAvgSub,
			float* combAvgSubtoOutBlock,
			unsigned int win_size)
{
	int thx=threadIdx.x;
	__shared__ float as_average;
	if (thx==0)
	{
		as_average = 0;
		for (int i = 0; i < win_size; i++)
			as_average += combSubMulToCombAvgSub[i];
		as_average /= win_size;
	}
	__syncthreads();
	if (thx < win_size)
		combAvgSubtoOutBlock[thx] = combSubMulToCombAvgSub[thx] - as_average;
}

__device__
void out_block_v2(float* reference,
					unsigned int sn,
					float* out_buffer,
					float* combAvgSubtoOutBlock,
					unsigned int win_size)
{
	unsigned int noz = blockIdx.x;
	// output
	/// output decoding
	reference[sn * N + noz] = out_buffer[noz*(BUFFER_SIZE+1)];
	//printf("%f ", out_buffer[noz*(BUFFER_SIZE+1)]);
	/// next state
	for (int i=0;i<BUFFER_SIZE;i++)
			out_buffer[noz*(BUFFER_SIZE+1)+i] = out_buffer[noz*(BUFFER_SIZE+1)+(i+1)];
	out_buffer[noz*(BUFFER_SIZE+1) + BUFFER_SIZE] = 0;
	unsigned int out_ignore_it = (BUFFER_SIZE - win_size)
			/ 2;
	for (int i = 0; i < BUFFER_SIZE - (2 * out_ignore_it); i++) {
		out_buffer[noz*(BUFFER_SIZE+1)+i + out_ignore_it] = out_buffer[noz*(BUFFER_SIZE+1) + i + out_ignore_it]
				+ combAvgSubtoOutBlock[i];

		//for (int i=0;i<out_buffer[noz].size();i++) printf("%f ", out_buffer[noz][i]); printf("\n");

	}
}

__global__
void computeNozzles_v2(float* reference,
						float* input_image,
						aoi* aoi_coordinates,
						float* parallelCoeffs,
						int* parallelSW,
						unsigned int image_height,
						unsigned int image_width,
						aoi* thread_state,
						float* image_parts,
						int* ac_sw,
						float* ac_samples,
						float* cc_coefs,
						float* out_buffer
)
{
	unsigned int noz = blockIdx.x;
	unsigned int thx = threadIdx.x;
	__shared__ int ac_ignore_it;
	__shared__ float imgPreproc2DDFA;
	__shared__ float autoCorrToCombSubMul[BUFFER_SIZE];
	__shared__ float ac_sampWin[(BUFFER_SIZE*2+1)];
	__shared__ float xCorrToCombSubMul[BUFFER_SIZE];
	__shared__ float combSubMulToCombAvgSub[BUFFER_SIZE];
	__shared__ float combAvgSubtoOutBlock[BUFFER_SIZE];

	for (unsigned int sn=0;sn<image_height;sn++)
	{
		// thread allocator
		thread_allocator_v2(input_image, aoi_coordinates, image_width, sn, thread_state, image_parts);

		// for all cuda blocks (nozzles):
		// pre-process image: inv and reduce
		imgPreproc2DDFA = preproc_image_v2(image_parts, image_width);

		// single DDFA
		/// auto correlation
		if (thx<BUFFER_SIZE)
		{
			autoCorrToCombSubMul[thx] = 0;
			ac_sampWin[thx] = 0;
			ac_sampWin[BUFFER_SIZE+thx] = 0;
		}
		if (thx==0)
		{
			ac_ignore_it = BUFFER_SIZE / 2 - ac_sw[noz];
			ac_sampWin[2*BUFFER_SIZE] = 0;
		}
		__syncthreads();
		auto_correlate_v2(imgPreproc2DDFA, parallelSW, sn, ac_samples, ac_sw, ac_ignore_it, ac_sampWin, autoCorrToCombSubMul);

		/// cross correlation
		/// note: we use the ac_samples, ac_ignore_it, ac_sampWin and ac_sw from the auto correlation stage
		if (thx<BUFFER_SIZE)
			xCorrToCombSubMul[thx] = 0;
		__syncthreads();
		//// output decoding
		cross_correlate_v2(ac_ignore_it, ac_sw, ac_sampWin, cc_coefs, parallelCoeffs, sn, xCorrToCombSubMul);

		// subtract and multiply ((x-y)/y)
		submul_v2(combSubMulToCombAvgSub, xCorrToCombSubMul, autoCorrToCombSubMul, ac_sw[noz] * 2);

		// average and subtract
		avgsub_v2(combSubMulToCombAvgSub, combAvgSubtoOutBlock, ac_sw[noz] * 2);

		// output
		/// output decoding
		out_block_v2(reference, sn, out_buffer, combAvgSubtoOutBlock, ac_sw[noz] * 2);
	}
	__syncthreads();
}

void compute_v2(float* reference,
				float* input_image,
				aoi* aoi_coordinates,
				float* parallelCoeffs,
				int* parallelSW,
				unsigned int image_height,
				unsigned int image_width)
{
	// allocate/transfer data on/to to the device
	float* d_reference;
	assert(hipSuccess == hipMalloc((void **) &d_reference, N*image_height * sizeof(float)));

	float* d_input_image;
	assert(hipSuccess == hipMalloc((void **) &d_input_image, image_width*image_height * sizeof(float)));
	hipMemcpy(d_input_image, input_image, image_width*image_height * sizeof(float), hipMemcpyHostToDevice);

	aoi* d_aoi_coordinates;
	assert(hipSuccess == hipMalloc((void **) &d_aoi_coordinates, N*image_height * sizeof(aoi)));
	hipMemcpy(d_aoi_coordinates, aoi_coordinates, N*image_height * sizeof(aoi), hipMemcpyHostToDevice);

	float* d_parallelCoeffs;
	assert(hipSuccess == hipMalloc((void **) &d_parallelCoeffs, BUFFER_SIZE*N*image_height * sizeof(float)));
	hipMemcpy(d_parallelCoeffs, parallelCoeffs, BUFFER_SIZE*N*image_height * sizeof(float), hipMemcpyHostToDevice);

	int* d_parallelSW;
	assert(hipSuccess == hipMalloc((void **) &d_parallelSW, N*image_height * sizeof(float)));
	hipMemcpy(d_parallelSW, parallelSW, N*image_height * sizeof(float), hipMemcpyHostToDevice);

	// state variables
	/// thread allocator
	aoi* thread_state;
	assert(hipSuccess == hipMalloc((void **) &thread_state, N * sizeof(aoi)));
	assert(hipSuccess == hipMemset((void*)thread_state, 0, N * sizeof(aoi)));
	float* image_parts;
	assert(hipSuccess == hipMalloc((void **) &image_parts, image_width * sizeof(float)));
	/// image preprocessor
	/// auto correlation
	float* ac_samples;
	assert(hipSuccess == hipMalloc((void **) &ac_samples, N * (BUFFER_SIZE + 1) * sizeof(float)));
	assert(hipSuccess == hipMemset((void*)ac_samples, 0, N * (BUFFER_SIZE + 1) * sizeof(float)));
	int* ac_sw;
	assert(hipSuccess == hipMalloc((void **) &ac_sw, N * sizeof(int)));
	assert(hipSuccess == hipMemset((void*)ac_sw, 0, N * sizeof(int)));
	/// cross correlation
	float* cc_coefs;
	assert(hipSuccess == hipMalloc((void **) &cc_coefs, N * (BUFFER_SIZE + 1) * sizeof(float)));
	assert(hipSuccess == hipMemset((void*)cc_coefs, 0, N * (BUFFER_SIZE + 1) * sizeof(float)));
	/// output block
	float* out_buffer;
	assert(hipSuccess == hipMalloc((void **) &out_buffer, N * (BUFFER_SIZE + 1) * sizeof(float)));
	assert(hipSuccess == hipMemset((void*)out_buffer, 0, N * (BUFFER_SIZE + 1) * sizeof(float)));

	// computation

	unsigned int cuda_threads = ((BUFFER_SIZE+1)/32+1)*32;
	computeNozzles_v2<<<N,cuda_threads>>>(d_reference, d_input_image, d_aoi_coordinates,
										d_parallelCoeffs, d_parallelSW, image_height, image_width,
										thread_state, image_parts, ac_sw, ac_samples,
										cc_coefs, out_buffer);

	hipMemcpy(reference, d_reference, N*image_height * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(thread_state);
	hipFree(image_parts);
	hipFree(ac_samples);
	hipFree(cc_coefs);
	hipFree(out_buffer);
	hipFree(ac_sw);

	hipFree(d_parallelSW);
	hipFree(d_parallelCoeffs);
	hipFree(d_aoi_coordinates);
	hipFree(d_input_image);
	hipFree(d_reference);
}

