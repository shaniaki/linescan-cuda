#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>

#include "globals.hpp"

////////////////////////////////////////////////////////////////////////////////
// export the callable function
extern "C"
void compute_v3(float* reference,
				float* input_image,
				aoi* aoi_coordinates,
				float* parallelCoeffs,
				int* parallelSW,
				unsigned int image_height,
				unsigned int image_width);

__device__
void thread_allocator_v3(float* input_image,
						aoi* aoi_coordinates,
						unsigned int image_width,
						unsigned int sn,
						aoi* thread_state,
						float* image_parts)
{
	// thread allocator
	unsigned int ta = blockIdx.x;
	/// output decoding
	int index = 0;
	for (int taod = thread_state->start;
			taod <= thread_state->end; taod++) {
		image_parts[(image_width / N) * ta + index] =
				input_image[(sn * image_width) + taod];
		index++;
	}
	image_parts[(image_width / N) * ta + index] = -1; // to terminate the image part
	/// next state
	if ((aoi_coordinates + sn * N + ta)->start != -1)
		*thread_state = aoi_coordinates[sn * N + ta];
}

__device__
float preproc_image_v3(float* image_parts, unsigned int image_width)
{
	unsigned int noz = blockIdx.x;
	float imgPreproc2DDFA = 0;
	float* imgpre = image_parts + (image_width / N) * noz;
	while (*(imgpre) != -1) {
		imgPreproc2DDFA += WHITE_VALUE - *(imgpre);
		imgpre++;
	}
	return imgPreproc2DDFA;
}

__device__
void auto_correlate_v3(float imgPreproc2DDFA,
						int* parallelSW,
						unsigned int sn,
						float* ac_samples,
						int* ac_sw,
						int ac_ignore_it,
						float* ac_sampWin,
						float* autoCorrToCombSubMul)
{
	unsigned int noz = blockIdx.x;
	unsigned int thx = threadIdx.x;
	//// output decoding
	if (thx<2*(*ac_sw))
		ac_sampWin[thx] = ac_samples[ac_ignore_it + thx];
	if (thx==0)
		ac_sampWin[2*(*ac_sw)] = ac_samples[ac_ignore_it + 2*(*ac_sw)];
	__syncthreads();
	if (thx<2* (*ac_sw))
		for (int c = 0; c <= (*ac_sw) * 2; c++) {
			int d = c + (thx-(*ac_sw)) + 1;
			int k = (thx-(*ac_sw)) + (*ac_sw);
			if ((d >= 0) && (d < (*ac_sw) * 2))
				autoCorrToCombSubMul[(*ac_sw) * 2 - k - 1] += ac_sampWin[c] * ac_sampWin[d];
		}
	__syncthreads();
	//// next state
	int ac_temp = parallelSW[sn * N + noz];
	if (ac_temp != -1)
		(*ac_sw) = ac_temp;

	__shared__ float ac_samples_temp[BUFFER_SIZE];
	if (thx<BUFFER_SIZE)
		ac_samples_temp[thx] = ac_samples[thx+1];
	__syncthreads();
	if (thx<BUFFER_SIZE)
		ac_samples[thx] = ac_samples_temp[thx];
	if (thx==0)
		ac_samples[BUFFER_SIZE] = imgPreproc2DDFA;
}

__device__
void cross_correlate_v3(int ac_ignore_it,
						int* ac_sw,
						float* ac_sampWin,
						float* cc_coefs,
						float* parallelCoeffs,
						unsigned int sn,
						float* xCorrToCombSubMul)
{
	unsigned int noz = blockIdx.x;
	unsigned int thx = threadIdx.x;
	//// output decoding
	__syncthreads();
	if (thx<2*(*ac_sw))
		for (int c = 0; c <= (*ac_sw) * 2; c++) {
			int d = c + (thx-(*ac_sw)) + 1;
			int k = (thx-(*ac_sw)) + (*ac_sw);
			if ((d >= 0) && (d < (*ac_sw) * 2))
				xCorrToCombSubMul[(*ac_sw) * 2 - k - 1] += ac_sampWin[c]
				                                                        * cc_coefs[ac_ignore_it + d];
		}
	__syncthreads();
	//// next state
	float* cc_temp = parallelCoeffs + sn * N * BUFFER_SIZE
			+ noz * BUFFER_SIZE;
	if (*cc_temp != -1)
		if (thx<BUFFER_SIZE)
				cc_coefs[thx] = cc_temp[thx];
}

__device__
void submul_v3(float* combSubMulToCombAvgSub,
			float* xCorrToCombSubMul,
			float* autoCorrToCombSubMul,
			unsigned int win_size)
{
	int i=threadIdx.x;
	if (i < win_size)
		combSubMulToCombAvgSub[i] = (xCorrToCombSubMul[i]
		                           - autoCorrToCombSubMul[i]) / autoCorrToCombSubMul[i];
}

__device__
void avgsub_v3(float* combSubMulToCombAvgSub,
			float* combAvgSubtoOutBlock,
			unsigned int win_size)
{
	int thx=threadIdx.x;
	__shared__ float as_average;
	if (thx==0)
	{
		as_average = 0;
		for (int i = 0; i < win_size; i++)
			as_average += combSubMulToCombAvgSub[i];
		as_average /= win_size;
	}
	__syncthreads();
	if (thx < win_size)
		combAvgSubtoOutBlock[thx] = combSubMulToCombAvgSub[thx] - as_average;
}

__device__
void out_block_v3(float* reference,
					unsigned int sn,
					float* out_buffer,
					float* combAvgSubtoOutBlock,
					unsigned int win_size)
{
	unsigned int noz = blockIdx.x;
	unsigned int thx = threadIdx.x;
	// output
	/// output decoding
	reference[sn * N + noz] = out_buffer[0];
	//printf("%f ", out_buffer[noz*(BUFFER_SIZE+1)]);
	/// next state
	__shared__ float out_buffer_temp[BUFFER_SIZE];
	if (thx<BUFFER_SIZE)
			out_buffer_temp[thx] = out_buffer[thx+1];
	__syncthreads();
	if (thx<BUFFER_SIZE)
		out_buffer[thx] = out_buffer_temp[thx];
	out_buffer[BUFFER_SIZE] = 0;
	unsigned int out_ignore_it = (BUFFER_SIZE - win_size)
			/ 2;
	if (thx < BUFFER_SIZE - (2 * out_ignore_it))
		out_buffer_temp[thx + out_ignore_it] = out_buffer[thx + out_ignore_it]
				+ combAvgSubtoOutBlock[thx];
	__syncthreads();
	if (thx < BUFFER_SIZE - (2 * out_ignore_it))
		out_buffer[thx+ out_ignore_it] = out_buffer_temp[thx+ out_ignore_it];
}

__global__
void computeNozzles_v3(float* reference,
						float* input_image,
						aoi* aoi_coordinates,
						float* parallelCoeffs,
						int* parallelSW,
						unsigned int image_height,
						unsigned int image_width,
						float* image_parts
)
{
	unsigned int thx = threadIdx.x;

	// state variables
	/// thread allocator
	__shared__ aoi thread_state;
	/// auto correlation
	__shared__ float ac_samples[BUFFER_SIZE + 1];
	__shared__ int ac_sw;
	/// cross correlation
	__shared__ float cc_coefs[BUFFER_SIZE + 1];
	/// output block
	__shared__ float out_buffer[BUFFER_SIZE + 1];
	//// initialization
	if (thx==0)
	{
		thread_state.start = 0;
		thread_state.end = 0;
		ac_sw = 0;
	}
	if (thx<BUFFER_SIZE+1)
	{
		ac_samples[thx] = 0;
		cc_coefs[thx] = 0;
		out_buffer[thx] = 0;
	}

	// inter-block communication
	__shared__ int ac_ignore_it;
	__shared__ float imgPreproc2DDFA;
	__shared__ float autoCorrToCombSubMul[BUFFER_SIZE];
	__shared__ float ac_sampWin[(BUFFER_SIZE*2+1)];
	__shared__ float xCorrToCombSubMul[BUFFER_SIZE];
	__shared__ float combSubMulToCombAvgSub[BUFFER_SIZE];
	__shared__ float combAvgSubtoOutBlock[BUFFER_SIZE];

	for (unsigned int sn=0;sn<image_height;sn++)
	{
		// thread allocator
		thread_allocator_v3(input_image, aoi_coordinates, image_width, sn, &thread_state, image_parts);

		// for all cuda blocks (nozzles):
		// pre-process image: inv and reduce
		imgPreproc2DDFA = preproc_image_v3(image_parts, image_width);

		// single DDFA
		/// auto correlation
		if (thx<BUFFER_SIZE)
		{
			autoCorrToCombSubMul[thx] = 0;
			ac_sampWin[thx] = 0;
			ac_sampWin[BUFFER_SIZE+thx] = 0;
		}
		if (thx==0)
		{
			ac_ignore_it = BUFFER_SIZE / 2 - ac_sw;
			ac_sampWin[2*BUFFER_SIZE] = 0;
		}

		__syncthreads();
		auto_correlate_v3(imgPreproc2DDFA, parallelSW, sn, ac_samples, &ac_sw, ac_ignore_it, ac_sampWin, autoCorrToCombSubMul);

		/// cross correlation
		/// note: we use the ac_samples, ac_ignore_it, ac_sampWin and ac_sw from the auto correlation stage
		if (thx<BUFFER_SIZE)
			xCorrToCombSubMul[thx] = 0;
		__syncthreads();
		//// output decoding
		cross_correlate_v3(ac_ignore_it, &ac_sw, ac_sampWin, cc_coefs, parallelCoeffs, sn, xCorrToCombSubMul);

		// subtract and multiply ((x-y)/y)
		submul_v3(combSubMulToCombAvgSub, xCorrToCombSubMul, autoCorrToCombSubMul, ac_sw * 2);

		// average and subtract
		avgsub_v3(combSubMulToCombAvgSub, combAvgSubtoOutBlock, ac_sw * 2);

		// output
		/// output decoding
		out_block_v3(reference, sn, out_buffer, combAvgSubtoOutBlock, ac_sw * 2);
	}
	__syncthreads();
}

void compute_v3(float* reference,
				float* input_image,
				aoi* aoi_coordinates,
				float* parallelCoeffs,
				int* parallelSW,
				unsigned int image_height,
				unsigned int image_width)
{
	// allocate/transfer data on/to to the device
	float* d_reference;
	assert(hipSuccess == hipMalloc((void **) &d_reference, N*image_height * sizeof(float)));

	float* d_input_image;
	assert(hipSuccess == hipMalloc((void **) &d_input_image, image_width*image_height * sizeof(float)));
	hipMemcpy(d_input_image, input_image, image_width*image_height * sizeof(float), hipMemcpyHostToDevice);

	aoi* d_aoi_coordinates;
	assert(hipSuccess == hipMalloc((void **) &d_aoi_coordinates, N*image_height * sizeof(aoi)));
	hipMemcpy(d_aoi_coordinates, aoi_coordinates, N*image_height * sizeof(aoi), hipMemcpyHostToDevice);

	float* d_parallelCoeffs;
	assert(hipSuccess == hipMalloc((void **) &d_parallelCoeffs, BUFFER_SIZE*N*image_height * sizeof(float)));
	hipMemcpy(d_parallelCoeffs, parallelCoeffs, BUFFER_SIZE*N*image_height * sizeof(float), hipMemcpyHostToDevice);

	int* d_parallelSW;
	assert(hipSuccess == hipMalloc((void **) &d_parallelSW, N*image_height * sizeof(float)));
	hipMemcpy(d_parallelSW, parallelSW, N*image_height * sizeof(float), hipMemcpyHostToDevice);


	float* image_parts;
	assert(hipSuccess == hipMalloc((void **) &image_parts, image_width * sizeof(float)));
	/// image preprocessor

	// computation

	unsigned int cuda_threads = ((BUFFER_SIZE+1)/32+1)*32;
	computeNozzles_v3<<<N,cuda_threads>>>(d_reference, d_input_image, d_aoi_coordinates,
										d_parallelCoeffs, d_parallelSW, image_height, image_width,
										image_parts);
	if ( hipSuccess != hipGetLastError() )
		printf( "Error in kernel call!\n" );

	assert(hipSuccess == hipMemcpy(reference, d_reference, N*image_height * sizeof(float), hipMemcpyDeviceToHost));

	hipFree(image_parts);

	hipFree(d_parallelSW);
	hipFree(d_parallelCoeffs);
	hipFree(d_aoi_coordinates);
	hipFree(d_input_image);
	hipFree(d_reference);
}

