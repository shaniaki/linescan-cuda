#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

#include "globals.hpp"


////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

extern "C"
float* readTiff(char* filename, unsigned int* w, unsigned int* h);

extern "C"
aoi* readAOIs(char* filename, unsigned int image_height, unsigned int num_threads);

extern "C"
float* readCoefs(char* filename, unsigned int image_height, unsigned int num_threads, unsigned int coefs_size);

extern "C"
int* readSWs(char* filename, unsigned int image_height, unsigned int num_threads);


extern "C"
void computeGold(float* reference,
				float* h_image_input,
				aoi* h_aoi_input,
				float* h_coeff_input,
				int* h_sw_input,
				unsigned int image_height,
				unsigned int image_width);

extern "C"
void compute_v1(float* reference,
				float* input_image,
				aoi* aoi_coordinates,
				float* parallelCoeffs,
				int* parallelSW,
				unsigned int image_height,
				unsigned int image_width);

extern "C"
void compute_v2(float* reference,
				float* input_image,
				aoi* aoi_coordinates,
				float* parallelCoeffs,
				int* parallelSW,
				unsigned int image_height,
				unsigned int image_width);

extern "C"
void compute_v3(float* reference,
				float* input_image,
				aoi* aoi_coordinates,
				float* parallelCoeffs,
				int* parallelSW,
				unsigned int image_height,
				unsigned int image_width);

/*
////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
testKernel(float *g_idata, float *g_odata)
{
    // shared memory
    // the size is determined by the host application
    extern  __shared__  float sdata[];

    // access thread id
    const unsigned int tid = threadIdx.x;
    // access number of threads in this block
    const unsigned int num_threads = blockDim.x;

    // read in input data from global memory
    sdata[tid] = g_idata[tid];
    __syncthreads();

    // perform some computations
    sdata[tid] = (float) num_threads * sdata[tid];
    __syncthreads();

    // write data to global memory
    g_odata[tid] = sdata[tid];
}
*/

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char **argv)
{
    runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char **argv)
{
    bool bTestResult = true;

    printf("%s Starting...\n\n", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    //int devID = findCudaDevice(argc, (const char **) (argv));

    unsigned int num_threads = N;
    float* h_coeff_input;
    float* h_image_input;
    int* h_sw_input;
    aoi* h_aoi_input;
    float *reference, *d_reference;
    unsigned int image_width, image_height;

    // read the input from the TIFF sizeof(aois) = inp_rows*image_row_width
    char input_name[20];
    sprintf(input_name,"%s/%d_input.tif",argv[1],DATASET);
	h_image_input = readTiff(input_name, &image_width, &image_height);

	// read the AOIs from the file sizeof(aois) = inp_rows*num_threads
	char aoi_name[20];
	sprintf(aoi_name,"%s/%d_aois.txt",argv[1],DATASET);
	h_aoi_input = readAOIs(aoi_name, image_height, num_threads);

	// read the coefs from the file sizeof(coefs) = inp_rows*num_threads*buffer_size
	char coefs_name[20];
	sprintf(coefs_name,"%s/%d_coefs.txt",argv[1],DATASET);
	h_coeff_input = readCoefs(coefs_name, image_height, num_threads, BUFFER_SIZE);

	// read the SWs from the file sizeof(sw) = inp_rows*num_threads
	char sws_name[20];
	sprintf(sws_name,"%s/%d_sws.txt",argv[1],DATASET);
	h_sw_input = readSWs(sws_name, image_height, num_threads);

	d_reference = (float*)malloc(image_height*num_threads*sizeof(float));

	double kernel_time=0;

	for (int i=0;i<=NUM_RUNS;i++)
	{
		struct timeval timerStart;
		gettimeofday(&timerStart, NULL);

		KERNELVER(d_reference, h_image_input, h_aoi_input, h_coeff_input, h_sw_input, image_height, image_width);

		struct timeval timerStop, timerElapsed;
		gettimeofday(&timerStop, NULL);
		timersub(&timerStop, &timerStart, &timerElapsed);
		if (i>0)
			kernel_time += timerElapsed.tv_sec*1000.0+timerElapsed.tv_usec/1000.0;
	}

	kernel_time /= NUM_RUNS;
	printf("Processing time: %f (ms)\n", kernel_time);

	// compute reference solution
	reference = (float*)malloc(image_height*num_threads*sizeof(float));
	computeGold(reference, h_image_input, h_aoi_input, h_coeff_input, h_sw_input, image_height, image_width);

	// check result
	for (int i=0;i<image_height;i++)
	{
		for (int j=0;j<N;j++)
		{
			if (abs(reference[i*N+j] - d_reference[i*N+j]) > 0.0001)
				printf("Error at image line %d for nozzle %d: reference=%f, calculated=%f\n", i, j, reference[i*N+j], d_reference[i*N+j]);
			//printf("%f ",d_reference[i*N+j]);
		}
		//printf("\n");
	}

    // cleanup memory
    free(h_image_input);
	free(h_aoi_input);
	free(h_coeff_input);
	free(h_sw_input);
	free(reference);
	free(d_reference);

    hipDeviceReset();
    exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
